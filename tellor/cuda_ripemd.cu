#include "hip/hip_runtime.h"
/*
 * sha256 + ripemd CUDA implementation for SM 5.0 GPUS (and lower)
 * Tanguy Pruvot and Provos Alexis - Jul / Sep 2016
 * Sponsored by TELLOR.IO team
 */

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>

#include <miner.h>

__global__ void tellor_difficulty_gpu(const uint64_t thread_count,
                                      uint32_t *d_hash,
                                      uint64_t *d_nonce);

__constant__ static uint32_t c_difficulty[8];

#if defined(__CUDA_ARCH__)
#include "cgbn/cgbn.h"

#define TPI 4

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, 256> env256_t;
typedef env256_t::cgbn_t cgbn_t;

__global__ void tellor_difficulty_gpu(const uint64_t thread_count,
                                      uint32_t *d_hash,
                                      uint64_t *d_nonce) {
  context_t bn_context;
  env256_t bn256_env(bn_context);
  cgbn_t hash, difficulty, remainder;

  const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
  if (thread >= thread_count) {
    return;
  }
  const uint32_t instance = thread / TPI;
  const uint32_t* input = &d_hash[instance<<3];
#if 0
  if (thread == 0) {
    for (int i = 0; i < 8; i++) {
    printf("%08x\n", c_difficulty[i]);
  }
  }
#endif
  
  cgbn_load(bn256_env, hash, (cgbn_mem_t<256>*)input);
  cgbn_load(bn256_env, difficulty, (cgbn_mem_t<256>*)&c_difficulty);
  cgbn_rem(bn256_env, remainder, hash, difficulty);

#if 0
  printf("%d %d: %08x mod %08x == %08x\n", instance, thread,
         cgbn_get_ui32(bn256_env, hash),
         cgbn_get_ui32(bn256_env, difficulty),
         cgbn_get_ui32(bn256_env, remainder));
#endif

if (cgbn_equals_ui32(bn256_env, remainder, 0)) {
    d_nonce[0] = instance;
}
  //  output[0] = (uint8_t)cgbn_equals_ui32(bn256_env, remainder, 0);
  //  cgbn_store(bn256_env, (cgbn_mem_t<256>*)&d_remainder[instance<<3], remainder);
}

#endif

#define TPI 4

__constant__  const uint32_t c_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};
__constant__ static uint32_t _ALIGN(16) c_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#ifdef __INTELLISENSE__
#define atomicExch(p,y) y
#define __byte_perm(x,y,c) x
#endif

// ------------------------------------------------------------------------------------------------

static const uint32_t cpu_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};

static const uint32_t cpu_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#define xor3b(a,b,c) (a ^ b ^ c)

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,2),ROTR32(x,13),ROTR32(x,22));
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,6),ROTR32(x,11),ROTR32(x,25));
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,7),ROTR32(x,18),(x>>3));
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,17),ROTR32(x,19),(x>>10));
}

__device__ __forceinline__ uint2 vectorizeswap(uint64_t v)
{
	uint2 result;
	asm("mov.b64 {%0,%1},%2; // vectorizeswap \n\t"
		: "=r"(result.y), "=r"(result.x) : "l"(v));
	return result;
}

#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define Ch(a, b, c)     (((b^c) & a) ^ c)

__device__ __forceinline__
static void sha2_step(const uint32_t a,const uint32_t b,const uint32_t c, uint32_t &d,const uint32_t e,const uint32_t f,const uint32_t g, uint32_t &h,const uint32_t in, const uint32_t Kshared)
{
	const uint32_t t1 = h + bsg2_1(e) + Ch(e, f, g) + Kshared + in;
	h = t1 + bsg2_0(a) + Maj(a, b, c);
	d+= t1;

}

__device__ __forceinline__
static void sha256_round_body(uint32_t *const __restrict__ in, uint32_t * __restrict__ state,const uint32_t *const __restrict__ Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

        sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	#pragma unroll 3
	for (uint32_t i=0; i<3; i++)
	{
		#pragma unroll 16
		for (uint32_t j = 0; j < 16; j++){
			in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
		}
		sha2_step(a, b, c, d, e, f, g, h, in[0], Kshared[16 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[1], Kshared[17 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[2], Kshared[18 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[3], Kshared[19 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[4], Kshared[20 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[5], Kshared[21 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[6], Kshared[22 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[7], Kshared[23 + 16 * i]);
		sha2_step(a, b, c, d, e, f, g, h, in[8], Kshared[24 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[9], Kshared[25 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

 __device__ __forceinline__
uint64_t cuda_swab64ll(const uint32_t x, const uint32_t y) {
	uint64_t r;
	asm("prmt.b32 %1, %1, 0, 0x0123; // swab64ll\n\t"
	    "prmt.b32 %2, %2, 0, 0x0123;\n\t"
	    "mov.b64 %0, {%1,%2};\n\t"
	  : "=l"(r): "r"(x), "r"(y) );
	return r;
}

__host__
void tellor_sha256_init(int thr_id)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_K), cpu_K, sizeof(cpu_K), 0, hipMemcpyHostToDevice);
}

__host__ void tellor_set_difficulty(const uint32_t *difficulty) {
  hipMemcpyToSymbol(HIP_SYMBOL(c_difficulty), difficulty, sizeof(uint32_t) * 8, 0,
                     hipMemcpyHostToDevice);
}

// ------------------------------------------------------------------------------------------

static __constant__ const uint32_t c_IV[5] = { 0x67452301u, 0xEFCDAB89u, 0x98BADCFEu, 0x10325476u, 0xC3D2E1F0u };

static __constant__ const uint32_t c_K1[5] = { 0, 0x5A827999, 0x6ED9EBA1, 0x8F1BBCDC, 0xA953FD4E };

static __constant__ const uint32_t c_K2[5] = { 0x50A28BE6, 0x5C4DD124, 0x6D703EF3, 0x7A6D76E9, 0 };

__device__ __forceinline__
static uint32_t ROTATE(const uint32_t x,const uint32_t r){
	if(r==8)
		return __byte_perm(x, 0, 0x2103);
	else
		return ROTL32(x,r);
}

/*
 * Round functions for RIPEMD-160.
 */
#define F1(x, y, z)   (x ^ y ^ z)
#define F2(x, y, z)   ((x & (y ^ z)) ^ z)
#define F3(x, y, z)   ((x | ~y) ^ z)
#define F4(x, y, z)   (y ^ ((x ^ y) & z))
#define F5(x, y, z)   (x ^ (y | ~z))

/*
 * Round constants for RIPEMD-160.
 */
#define RR(a, b, c, d, e, f, s, r, k) { \
	a = e + ROTATE((a + r + k + f(b, c, d)), s); \
	c = ROTL32(c, 10); \
}

#define ROUND1(a, b, c, d, e, f, s, r, k) \
	RR(a[0], b[0], c[0], d[0], e[0], f, s, r, c_K1[k])

#define ROUND2(a, b, c, d, e, f, s, r, k) \
	RR(a[1], b[1], c[1], d[1], e[1], f, s, r, c_K2[k])

#define RIPEMD160_ROUND_BODY(in, h) { \
	uint32_t A[2], B[2], C[2], D[2], E[2]; \
	uint32_t tmp; \
\
	A[0] = A[1] = h[0]; \
	B[0] = B[1] = h[1]; \
	C[0] = C[1] = h[2]; \
	D[0] = D[1] = h[3]; \
	E[0] = E[1] = h[4]; \
\
	ROUND1(A, B, C, D, E, F1, 11, in[ 0],  0); \
	ROUND1(E, A, B, C, D, F1, 14, in[ 1],  0); \
	ROUND1(D, E, A, B, C, F1, 15, in[ 2],  0); \
	ROUND1(C, D, E, A, B, F1, 12, in[ 3],  0); \
	ROUND1(B, C, D, E, A, F1,  5, in[ 4],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[ 5],  0); \
	ROUND1(E, A, B, C, D, F1,  7, in[ 6],  0); \
	ROUND1(D, E, A, B, C, F1,  9, in[ 7],  0); \
	ROUND1(C, D, E, A, B, F1, 11, in[ 8],  0); \
	ROUND1(B, C, D, E, A, F1, 13, in[ 9],  0); \
	ROUND1(A, B, C, D, E, F1, 14, in[10],  0); \
	ROUND1(E, A, B, C, D, F1, 15, in[11],  0); \
	ROUND1(D, E, A, B, C, F1,  6, in[12],  0); \
	ROUND1(C, D, E, A, B, F1,  7, in[13],  0); \
	ROUND1(B, C, D, E, A, F1,  9, in[14],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[15],  0); \
\
	ROUND1(E, A, B, C, D, F2,  7, in[ 7],  1); \
	ROUND1(D, E, A, B, C, F2,  6, in[ 4],  1); \
	ROUND1(C, D, E, A, B, F2,  8, in[13],  1); \
	ROUND1(B, C, D, E, A, F2, 13, in[ 1],  1); \
	ROUND1(A, B, C, D, E, F2, 11, in[10],  1); \
	ROUND1(E, A, B, C, D, F2,  9, in[ 6],  1); \
	ROUND1(D, E, A, B, C, F2,  7, in[15],  1); \
	ROUND1(C, D, E, A, B, F2, 15, in[ 3],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[12],  1); \
	ROUND1(A, B, C, D, E, F2, 12, in[ 0],  1); \
	ROUND1(E, A, B, C, D, F2, 15, in[ 9],  1); \
	ROUND1(D, E, A, B, C, F2,  9, in[ 5],  1); \
	ROUND1(C, D, E, A, B, F2, 11, in[ 2],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[14],  1); \
	ROUND1(A, B, C, D, E, F2, 13, in[11],  1); \
	ROUND1(E, A, B, C, D, F2, 12, in[ 8],  1); \
\
	ROUND1(D, E, A, B, C, F3, 11, in[ 3],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[10],  2); \
	ROUND1(B, C, D, E, A, F3,  6, in[14],  2); \
	ROUND1(A, B, C, D, E, F3,  7, in[ 4],  2); \
	ROUND1(E, A, B, C, D, F3, 14, in[ 9],  2); \
	ROUND1(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[ 8],  2); \
	ROUND1(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND1(A, B, C, D, E, F3, 14, in[ 2],  2); \
	ROUND1(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND1(D, E, A, B, C, F3, 13, in[ 0],  2); \
	ROUND1(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND1(B, C, D, E, A, F3,  5, in[13],  2); \
	ROUND1(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND1(E, A, B, C, D, F3,  7, in[ 5],  2); \
	ROUND1(D, E, A, B, C, F3,  5, in[12],  2); \
\
	ROUND1(C, D, E, A, B, F4, 11, in[ 1],  3); \
	ROUND1(B, C, D, E, A, F4, 12, in[ 9],  3); \
	ROUND1(A, B, C, D, E, F4, 14, in[11],  3); \
	ROUND1(E, A, B, C, D, F4, 15, in[10],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 0],  3); \
	ROUND1(C, D, E, A, B, F4, 15, in[ 8],  3); \
	ROUND1(B, C, D, E, A, F4,  9, in[12],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[ 4],  3); \
	ROUND1(E, A, B, C, D, F4,  9, in[13],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 3],  3); \
	ROUND1(C, D, E, A, B, F4,  5, in[ 7],  3); \
	ROUND1(B, C, D, E, A, F4,  6, in[15],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[14],  3); \
	ROUND1(E, A, B, C, D, F4,  6, in[ 5],  3); \
	ROUND1(D, E, A, B, C, F4,  5, in[ 6],  3); \
	ROUND1(C, D, E, A, B, F4, 12, in[ 2],  3); \
\
	ROUND1(B, C, D, E, A, F5,  9, in[ 4],  4); \
	ROUND1(A, B, C, D, E, F5, 15, in[ 0],  4); \
	ROUND1(E, A, B, C, D, F5,  5, in[ 5],  4); \
	ROUND1(D, E, A, B, C, F5, 11, in[ 9],  4); \
	ROUND1(C, D, E, A, B, F5,  6, in[ 7],  4); \
	ROUND1(B, C, D, E, A, F5,  8, in[12],  4); \
	ROUND1(A, B, C, D, E, F5, 13, in[ 2],  4); \
	ROUND1(E, A, B, C, D, F5, 12, in[10],  4); \
	ROUND1(D, E, A, B, C, F5,  5, in[14],  4); \
	ROUND1(C, D, E, A, B, F5, 12, in[ 1],  4); \
	ROUND1(B, C, D, E, A, F5, 13, in[ 3],  4); \
	ROUND1(A, B, C, D, E, F5, 14, in[ 8],  4); \
	ROUND1(E, A, B, C, D, F5, 11, in[11],  4); \
	ROUND1(D, E, A, B, C, F5,  8, in[ 6],  4); \
	ROUND1(C, D, E, A, B, F5,  5, in[15],  4); \
	ROUND1(B, C, D, E, A, F5,  6, in[13],  4); \
\
	ROUND2(A, B, C, D, E, F5,  8, in[ 5],  0); \
	ROUND2(E, A, B, C, D, F5,  9, in[14],  0); \
	ROUND2(D, E, A, B, C, F5,  9, in[ 7],  0); \
	ROUND2(C, D, E, A, B, F5, 11, in[ 0],  0); \
	ROUND2(B, C, D, E, A, F5, 13, in[ 9],  0); \
	ROUND2(A, B, C, D, E, F5, 15, in[ 2],  0); \
	ROUND2(E, A, B, C, D, F5, 15, in[11],  0); \
	ROUND2(D, E, A, B, C, F5,  5, in[ 4],  0); \
	ROUND2(C, D, E, A, B, F5,  7, in[13],  0); \
	ROUND2(B, C, D, E, A, F5,  7, in[ 6],  0); \
	ROUND2(A, B, C, D, E, F5,  8, in[15],  0); \
	ROUND2(E, A, B, C, D, F5, 11, in[ 8],  0); \
	ROUND2(D, E, A, B, C, F5, 14, in[ 1],  0); \
	ROUND2(C, D, E, A, B, F5, 14, in[10],  0); \
	ROUND2(B, C, D, E, A, F5, 12, in[ 3],  0); \
	ROUND2(A, B, C, D, E, F5,  6, in[12],  0); \
\
	ROUND2(E, A, B, C, D, F4,  9, in[ 6],  1); \
	ROUND2(D, E, A, B, C, F4, 13, in[11],  1); \
	ROUND2(C, D, E, A, B, F4, 15, in[ 3],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[ 7],  1); \
	ROUND2(A, B, C, D, E, F4, 12, in[ 0],  1); \
	ROUND2(E, A, B, C, D, F4,  8, in[13],  1); \
	ROUND2(D, E, A, B, C, F4,  9, in[ 5],  1); \
	ROUND2(C, D, E, A, B, F4, 11, in[10],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[14],  1); \
	ROUND2(A, B, C, D, E, F4,  7, in[15],  1); \
	ROUND2(E, A, B, C, D, F4, 12, in[ 8],  1); \
	ROUND2(D, E, A, B, C, F4,  7, in[12],  1); \
	ROUND2(C, D, E, A, B, F4,  6, in[ 4],  1); \
	ROUND2(B, C, D, E, A, F4, 15, in[ 9],  1); \
	ROUND2(A, B, C, D, E, F4, 13, in[ 1],  1); \
	ROUND2(E, A, B, C, D, F4, 11, in[ 2],  1); \
\
	ROUND2(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND2(C, D, E, A, B, F3,  7, in[ 5],  2); \
	ROUND2(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND2(A, B, C, D, E, F3, 11, in[ 3],  2); \
	ROUND2(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND2(D, E, A, B, C, F3,  6, in[14],  2); \
	ROUND2(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND2(B, C, D, E, A, F3, 14, in[ 9],  2); \
	ROUND2(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND2(E, A, B, C, D, F3, 13, in[ 8],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[12],  2); \
	ROUND2(C, D, E, A, B, F3, 14, in[ 2],  2); \
	ROUND2(B, C, D, E, A, F3, 13, in[10],  2); \
	ROUND2(A, B, C, D, E, F3, 13, in[ 0],  2); \
	ROUND2(E, A, B, C, D, F3,  7, in[ 4],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[13],  2); \
\
	ROUND2(C, D, E, A, B, F2, 15, in[ 8],  3); \
	ROUND2(B, C, D, E, A, F2,  5, in[ 6],  3); \
	ROUND2(A, B, C, D, E, F2,  8, in[ 4],  3); \
	ROUND2(E, A, B, C, D, F2, 11, in[ 1],  3); \
	ROUND2(D, E, A, B, C, F2, 14, in[ 3],  3); \
	ROUND2(C, D, E, A, B, F2, 14, in[11],  3); \
	ROUND2(B, C, D, E, A, F2,  6, in[15],  3); \
	ROUND2(A, B, C, D, E, F2, 14, in[ 0],  3); \
	ROUND2(E, A, B, C, D, F2,  6, in[ 5],  3); \
	ROUND2(D, E, A, B, C, F2,  9, in[12],  3); \
	ROUND2(C, D, E, A, B, F2, 12, in[ 2],  3); \
	ROUND2(B, C, D, E, A, F2,  9, in[13],  3); \
	ROUND2(A, B, C, D, E, F2, 12, in[ 9],  3); \
	ROUND2(E, A, B, C, D, F2,  5, in[ 7],  3); \
	ROUND2(D, E, A, B, C, F2, 15, in[10],  3); \
	ROUND2(C, D, E, A, B, F2,  8, in[14],  3); \
\
	ROUND2(B, C, D, E, A, F1,  8, in[12],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[15],  4); \
	ROUND2(E, A, B, C, D, F1, 12, in[10],  4); \
	ROUND2(D, E, A, B, C, F1,  9, in[ 4],  4); \
	ROUND2(C, D, E, A, B, F1, 12, in[ 1],  4); \
	ROUND2(B, C, D, E, A, F1,  5, in[ 5],  4); \
	ROUND2(A, B, C, D, E, F1, 14, in[ 8],  4); \
	ROUND2(E, A, B, C, D, F1,  6, in[ 7],  4); \
	ROUND2(D, E, A, B, C, F1,  8, in[ 6],  4); \
	ROUND2(C, D, E, A, B, F1, 13, in[ 2],  4); \
	ROUND2(B, C, D, E, A, F1,  6, in[13],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[14],  4); \
	ROUND2(E, A, B, C, D, F1, 15, in[ 0],  4); \
	ROUND2(D, E, A, B, C, F1, 13, in[ 3],  4); \
	ROUND2(C, D, E, A, B, F1, 11, in[ 9],  4); \
	ROUND2(B, C, D, E, A, F1, 11, in[11],  4); \
\
	tmp  = h[1] + C[0] + D[1]; \
	h[1] = h[2] + D[0] + E[1]; \
	h[2] = h[3] + E[0] + A[1]; \
	h[3] = h[4] + A[0] + B[1]; \
	h[4] = h[0] + B[0] + C[1]; \
	h[0] = tmp; \
}

__global__
#if CUDA_VERSION > 6050
__launch_bounds__(1024,2) /* to force 32 regs */
#endif
void tellor_ripemd(const uint64_t threads, uint32_t *Hash256){
	uint32_t dat[16];
	uint32_t h[5];
	uint32_t buf[8]; // align for vectorize

        int thread = blockIdx.x * blockDim.x + threadIdx.x;
	if(thread<threads){

		uint32_t* input = (uint32_t*) (&Hash256[thread<<3]);

		*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[0]);

		dat[8] = 0x80;

		#pragma unroll 7
		for (int i=9;i<16;i++) dat[i] = 0;

		dat[14] = 0x100; // size in bits

		#pragma unroll 5
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

                RIPEMD160_ROUND_BODY(dat, h);

                //		#pragma unroll 5
                //		for (int i=0; i<5; i++)
                //			buf[i] = h[i];

		// second 32 bytes block hash

                //		*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[8]);

                //		dat[8] = 0x80;

                //		#pragma unroll 7
                //		for (int i=9;i<16;i++) dat[i] = 0;

                //		dat[14] = 0x100; // size in bits

                //		#pragma unroll 5
                //		for (int i=0; i<5; i++)
                //			h[i] = c_IV[i];

                //		RIPEMD160_ROUND_BODY(dat, h);

		#pragma unroll 5
		for (int i=0;i<5;i++) dat[i] = h[i];
                //		#pragma unroll 5
                //		for (int i=0;i<5;i++) dat[i+5] = cuda_swab32(h[i]);

		*(uint2x4*)&input[ 0] = *(uint2x4*)&dat[ 0];
                //		*(uint2*)&input[ 8] = *(uint2*)&dat[ 8];
#if 0
                printf("GPU RIPEMD HASH:\n");
                for (int i = 0; i < 5; i++) {
                  printf("%08x ", Hash256[i]);
                }
                printf("\n");
#endif
	}
}

__global__
#if CUDA_VERSION > 6050
__launch_bounds__(768,2) /* to force 32 regs */
#endif
    void tellor_sha256_gpu_hash_final(const uint64_t threads, uint32_t *Hash256)
{
	const uint64_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t dat[16] = {0};
	uint32_t buf[8]; // align for vectorize
	if (thread < threads)
	{

          uint32_t* input = (uint32_t*) (&Hash256[thread<<3]);
          //*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[0]);
          //          *(uint2*)&dat[8] = __ldg((uint2*)&input[8]);
#pragma unroll 5
          for (int i = 0; i < 5; i++) {
            dat[i] = cuda_swab32(input[i]);
          }
          dat[5] = 0x80000000;
#pragma unroll 10
          for (int i=6; i<15; i++) dat[i] = 0;
          dat[15] = 0xa0;

          *(uint2x4*)&buf[0] = *(uint2x4*)&c_H256[0];

          sha256_round_body(dat, buf, c_K); // s_K uses too many regs
          //          *(uint2x4*)&Hash256[thread<<3] = *(uint2x4*)&buf[0];

          //          printf("GPU SHA256 HASH:\n");
                #pragma unroll 8
                for (int i = 0; i < 8; i++) {
                  input[i] = (buf[7-i]);
                  //                  printf("%08x ", input[i]);
                }
                //                printf("\n");
	}
}

__host__
    void tellor_sha256_hash_final(int thr_id, uint64_t threads, uint32_t *d_inputHash)
{
	int threadsperblock;

	threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

        tellor_sha256_gpu_hash_final <<<grid, block>>> (threads, (uint32_t*) d_inputHash);
        //tellor_sha256_gpu_hash_final <<<4,1>>> (threads, (uint32_t*) d_inputHash);
}

__host__
    void tellor_ripemd_hash(int thr_id, uint64_t threads, uint32_t *d_inputHash)
{
	int threadsperblock;

	threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

        tellor_ripemd <<<grid, block>>> (threads,(uint32_t*) d_inputHash);
        //tellor_ripemd <<<4,1>>> (threads,(uint32_t*) d_inputHash);
}

__host__ void tellor_difficulty(int gpu_id, uint64_t thread_count, uint32_t* d_hash,
                                uint64_t *d_nonce) {
  int threadsperblock = 512;
  int ipb = threadsperblock/TPI;
  dim3 grid((thread_count + ipb - 1) / threadsperblock);
  dim3 block(threadsperblock);
  
  tellor_difficulty_gpu <<<grid, block>>> (thread_count, d_hash, d_nonce);
  //tellor_difficulty_gpu<<<4,4>>>(thread_count, d_hash, d_nonce);
}
