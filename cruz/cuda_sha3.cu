#include "hip/hip_runtime.h"
#include "miner.h"

extern "C" {
#include <stdint.h>
#include <memory.h>
}

#include "cuda_helper.h"

static const uint64_t host_sha3_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

static uint64_t *d_KNonce[MAX_GPUS];

__constant__ uint32_t pTarget[2];
__constant__ uint64_t sha3_round_constants[24];
__constant__ uint64_t c_PaddedMessage80[25]; // padded message (80 bytes + padding?)

__device__ __forceinline__
static void sha3_blockv30(uint64_t *s, const uint64_t *sha3_round_constants)
{
	size_t i;
	uint64_t t[5], u[5], v, w;

	/* absorb input */

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= sha3_round_constants[i];
	}
}

__device__
char* format_nonce(uint64_t nonce, char* buffer) { 
  const char digits[] =
      "0001020304050607080910111213141516171819"
      "2021222324252627282930313233343536373839"
      "4041424344454647484950515253545556575859"
      "6061626364656667686970717273747576777879"
      "8081828384858687888990919293949596979899";
  char* position = buffer + 19;
  while (nonce >= 100) {
    unsigned index = static_cast<unsigned>((nonce % 100) * 2);
    nonce /= 100;
    *--position = digits[index + 1];
    *--position = digits[index];
  }
  if (nonce < 10) {
    *--position = static_cast<char>('0' + nonce);
    return position;
  }
  unsigned index = static_cast<unsigned>(nonce * 2);
  *--position = digits[index + 1];
  *--position = digits[index];
  return position;
} 

__global__
void cruz_gpu_hash(uint32_t threads, uint64_t startNounce, uint64_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
          uint64_t nounce = startNounce + thread + 1000000000000000;
		uint64_t sha3_gpu_state[25];
#pragma unroll 25
		for (int i=0; i<25; i++) {
                  sha3_gpu_state[i] = c_PaddedMessage80[i];
		}
                //                uint64_t n = 6372506688733637;
                //                printf("format nonce: %s\n", format_nonce(n));
                
                //                char* nonce = "6372506688733637";
                char buffer[20];
                char* nonce = format_nonce(nounce, buffer);
                //                if (nounce % (65536 * 4) == 0) {
                //                                  printf("%s\n", nonce);
                //                }
#pragma unroll 16
                for (int i = 0; i < 16; i++) {
                  ((uint8_t*)sha3_gpu_state)[73 + i] ^= nonce[i];
                }
                //                memcpy(&((uint8_t*)sha3_gpu_state)[73], nonce, 16);
		sha3_blockv30(sha3_gpu_state, sha3_round_constants);
                //                for (int i = 0; i < 8; i++) {
                //                  printf("%08x ", pTarget[i]);
                //                }
                //                  printf("%016lx %016lx (%" PRIu64 ")\n",
                //                         (sha3_gpu_state)[0],
                //                (sha3_gpu_state)[1],
                //                         nounce);
		if (cuda_swab32(((uint32_t*)sha3_gpu_state)[0]) <= pTarget[0] &&
                    cuda_swab32(((uint32_t*)sha3_gpu_state)[1]) <= pTarget[1]) {
                  //                  printf("nonce: %s\n", nonce);
                  //                  printf("%08x %08x (%08x %08x) (%" PRIu64 ")\n",
                  //                         ((uint32_t*)sha3_gpu_state)[0],
                  //                         ((uint32_t*)sha3_gpu_state)[1],
                  //                         pTarget[0], pTarget[1],
                  //                         nounce);
                  resNounce[0] = nounce;
                }
	}
}

__host__
void cruz_cpu_hash(int thr_id, uint32_t threads, uint64_t startNounce, uint64_t *resNonces, int order)
{
	hipMemset(d_KNonce[thr_id], 0xff, sizeof(uint64_t));
	const uint32_t threadsperblock = 128;

        dim3 grid((threads + threadsperblock-1)/threadsperblock);
        dim3 block(threadsperblock);
        //dim3 grid = 1;
        //dim3 block = 1;

	size_t shared_size = 0;

        cruz_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, d_KNonce[thr_id]);
        CUDA_SAFE_CALL(hipMemcpy(resNonces, d_KNonce[thr_id], sizeof(uint64_t), hipMemcpyDeviceToHost));
        // 	hipDeviceSynchronize();
}

#if 0
__global__ __launch_bounds__(256,3)
void sha3256_sm3_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
#if __CUDA_ARCH__ >= 350 /* tpr: to double check if faster on SM5+ */
		uint2 sha3_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4) sha3_gpu_state[i] = vectorize(outputHash[i*threads+thread]);
			else     sha3_gpu_state[i] = make_uint2(0, 0);
		}
		sha3_gpu_state[4]  = make_uint2(6, 0);
		sha3_gpu_state[16] = make_uint2(0, 0x80000000);
		sha3_blockv35(sha3_gpu_state, sha3_round_constants);

		#pragma unroll 4
		for (int i=0; i<4; i++)
			outputHash[i*threads+thread] = devectorize(sha3_gpu_state[i]);
#else
		uint64_t sha3_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4)
				sha3_gpu_state[i] = outputHash[i*threads+thread];
			else
				sha3_gpu_state[i] = 0;
		}
		sha3_gpu_state[4]  = 0x0000000000000006;
		sha3_gpu_state[16] = 0x8000000000000000;

		sha3_blockv30(sha3_gpu_state, sha3_round_constants);
		#pragma unroll 4
		for (int i = 0; i<4; i++)
			outputHash[i*threads + thread] = sha3_gpu_state[i];
#endif
	}
}

__host__
void cruz_sm3_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	sha3256_sm3_gpu_hash_32 <<<grid, block>>> (threads, startNounce, d_outputHash);
	MyStreamSynchronize(NULL, order, thr_id);
}
#endif

__host__
void sha3_keccakf_cu(uint64_t st[25]) {
  // constants
  const uint64_t keccakf_rndc[24] = {
      0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
      0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
      0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
      0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
      0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
      0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
      0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
      0x8000000000008080, 0x0000000080000001, 0x8000000080008008};
  const int keccakf_rotc[24] = {1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14,
                                27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44};
  const int keccakf_piln[24] = {10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4,
                                15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1};

  // variables
  int i, j, r;
  uint64_t t, bc[5];

#if 0
#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
  uint8_t *v;

  // endianess conversion. this is redundant on little-endian targets
  for (i = 0; i < 25; i++) {
    v = (uint8_t *)&st[i];
    st[i] = ((uint64_t)v[0]) | (((uint64_t)v[1]) << 8) |
            (((uint64_t)v[2]) << 16) | (((uint64_t)v[3]) << 24) |
            (((uint64_t)v[4]) << 32) | (((uint64_t)v[5]) << 40) |
            (((uint64_t)v[6]) << 48) | (((uint64_t)v[7]) << 56);
  }
#endif
#endif

  // actual iteration
  for (r = 0; r < 24; r++) {

    // Theta
    for (i = 0; i < 5; i++)
      bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

    for (i = 0; i < 5; i++) {
      t = bc[(i + 4) % 5] ^ ROTL64(bc[(i + 1) % 5], 1);
      for (j = 0; j < 25; j += 5)
        st[j + i] ^= t;
    }

    // Rho Pi
    t = st[1];
    for (i = 0; i < 24; i++) {
      j = keccakf_piln[i];
      bc[0] = st[j];
      st[j] = ROTL64(t, keccakf_rotc[i]);
      t = bc[0];
    }

    //  Chi
    for (j = 0; j < 25; j += 5) {
      for (i = 0; i < 5; i++)
        bc[i] = st[j + i];
      for (i = 0; i < 5; i++)
        st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
    }

    //  Iota
    st[0] ^= keccakf_rndc[r];
  }

#if 0
#if __BYTE_ORDER__ != __ORDER_LITTLE_ENDIAN__
  // endianess conversion. this is redundant on little-endian targets
  for (i = 0; i < 25; i++) {
    v = (uint8_t *)&st[i];
    t = st[i];
    v[0] = t & 0xFF;
    v[1] = (t >> 8) & 0xFF;
    v[2] = (t >> 16) & 0xFF;
    v[3] = (t >> 24) & 0xFF;
    v[4] = (t >> 32) & 0xFF;
    v[5] = (t >> 40) & 0xFF;
    v[6] = (t >> 48) & 0xFF;
    v[7] = (t >> 56) & 0xFF;
  }
#endif
#endif
}

__host__
static void sha3_block(uint64_t *s, const uint64_t *sha3_round_constants)
{
	size_t i;
	uint64_t t[5], u[5], v, w;

	/* absorb input */

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= sha3_round_constants[i];
	}
}

__host__
void cruz_setBlock_345(uint64_t* data, const void *pTargetIn, size_t block_size) {
  uint64_t state[25] = {0};
  
#if 0
  uint64_t state[25] = {
    0x59830c7b2e360638, 0x25c8054aa1fe772b, 0xa63a7afb2adab057, 0x8bc6407a70705868,
    0x2ab5a20f0f258c5a, 0x50f42c70cc6b62ab, 0xee7a078b1654c33a, 0x0995782d81eef31e,
    0xf9f8765b14dbfbf1, 0x4c0152ba68c7c487, 0xa2d4c455c5cc1b4c, 0x74427549fe2e0263,
    0x74ff6836e434a5d7, 0x11c2fb9ac37cebcf, 0x0c088eff2caad46b, 0x1d4c9465b20a030e,
    0xf2d74572132d0014, 0x3a93d6c7d3a14db2, 0x9eaa277d7c920f16, 0x683a03db87cfa81c,
    0x05ce9391364f79f0, 0xac50db98756c54f5, 0x0539ab0cd4f1c619, 0xcd346a07f393ffba,
    0xefacb5358cb691f6};
  #endif

  for (int i = 0; i < 17; i++) {
    state[i] = data[i];
  }
  //  printf("**** UPDATE:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");
  //  sha3_block(state, sha3_round_constants);
  sha3_keccakf_cu(state);
  //  printf("**** STATE:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");

  for (int i = 0; i < 17; i++) {
    state[i] ^= data[17 + i];
  }
  //  printf("**** UPDATE:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");
  sha3_keccakf_cu(state);
  //  sha3_block(state, sha3_round_constants);

  //  printf("**** STATE:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");

  for (int i = 0; i < 73; i++) {
    ((uint8_t*)state)[i] ^= ((uint8_t*)data)[272 + i];
  }
  for (int i = 89; i < block_size; i++) {
    ((uint8_t*)state)[i] ^= ((uint8_t*)data)[272 + i];
  }
  ((uint8_t*)state)[block_size] ^= 0x06;
  ((uint8_t*)state)[135] ^= 0x80;
  //  printf("**** UPDATE:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");
  //  sha3_block(state, sha3_round_constants);
  // sha3_keccakf_cu(state);

  //  printf("**** FINAL:\n");
  //  for (int i = 0; i < 25; i++) {
  //    printf("%016lx ", state[i]);
  //  }
  //  printf("\n");

  
  //  unsigned char PaddedMessage[80];
  //	memcpy(PaddedMessage, pdata, 80);
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 2*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), state, 25*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void cruz_sm3_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sha3_round_constants), host_sha3_round_constants,
				sizeof(host_sha3_round_constants), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc(&d_KNonce[thr_id], sizeof(uint64_t)));
}

__host__
void cruz_sm3_free(int thr_id)
{
	hipFree(d_KNonce[thr_id]);
}
